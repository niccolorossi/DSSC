
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N (2048*2048)
#define THREAD_PER_BLOCK 512

__global__ void reverse( int *a, int *b ) {
       int index=threadIdx.x+blockIdx.x*blockDim.x;
       b[index] = a[N-index];
}

void random_ints(int *p, int n) {
	int i;
	for(i=0; i<n; i++) {
		p[i]=rand();
	}
}

int main( void ) {
    int *a, *b, *c;               // host copies of a, b, c
    int *dev_a, *dev_b;           // device copies of a, b, c
    int size = N * sizeof( int ); // we need space for N   									// integers
    int i;

    // allocate device copies of a, b, c
    hipMalloc( (void**)&dev_a, size );
    hipMalloc( (void**)&dev_b, size );

    a = (int*)malloc( size ); 
    b = (int*)malloc( size );
    c = (int*)malloc( size );

    random_ints( a, N ); 
    // copy inputs to device
    hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice );

    // launch an add() kernel with N threads
    reverse<<< N/THREAD_PER_BLOCK, THREAD_PER_BLOCK >>>( dev_a, dev_b);

    // copy device result back to host copy of c
    hipMemcpy( b, dev_b, size,   hipMemcpyDeviceToHost );

    for(i=0; i<N; i++) {
	c[i]=a[N-i];
	if(b[i]!=c[i]) {
	   printf("error: expected %d, got %d!\n",c[i], b[i]);
	   break;
	}
    }
    if(i==N) {
	   printf("correct!\n");
    }    

 
    free( a ); free( b ); free( c );
    hipFree( dev_a );
    hipFree( dev_b );
    return 0;
}
