#include "hip/hip_runtime.h"
// transpose in C

for(i=0; i<N; i++) {
  for(j=0; j<N; j++) {
    mat_in[i][j]=mat_out[j][i];
  }
 }

// parallelize in C

/*
#pragma omp for private (i,j) 
for(i=0; i<N; i++) {
  for(j=i; j<N; j++) {
  }
}
*/

///////////////////////////////////////////

#define NTHS 1024





// transpose in CUDA

// these pointers store memory for input and output matrices, for host and device
double *MAT_IN_h, *MAT_OUT_h, *MAT_IN_D, *MAT_OUT_D;

MAT_IN_h = (double *) malloc(N*N*sizeof(double));
MAT_OUT_h = //same as above;

hipMalloc(&MAT_IN_d, N*N*sizeof(double));
hipMalloc(&MAT_OUT_d, N*N*sizeof(double));

INIT(MAT_IN_h); // function that initialises host matrix (in CPU)

hipMemcpy(MAT_IN_h, MAT_IN_d, N*N*sizeof(int)); // copying memory into device

MAT_TRANSP<<<N_BLOCKS, N_Threads>>(MAT_IN_d, MAT_OUT_d); //1024 max number of threads in a GPU (see #define above)
// you can define
N_BLOCKS = N*N/NTHS;
// blocks enclose portions of matrix 
// they ecompass portions of lines...

Idx = ThreadIdx.x + blockIdx.x*blockDim.x;
//aux variables

x = Idx%N;
y = Idx/N;

mat[y+N*x] = mat[x+N*y]; //or something like that

// another possible solution
// seems like we already have two dimensions; block idx and thread idx

MAT_TRANSP<<N,1024>> (MAT_IN_D, MAT_OUT_D)

//kernel definition

__global__ mat_transp(double * M.in, double * M.out) {
	   y = blockIdx.x;
	   x = threadIdx.x;
	   while(x<N) {

	   	   M.out(y*N +x) = M.in(x*n+y);
	  	   x+=blockDim.x;
		   }


}




// matrix multiplication






