
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 64
#define NUM_BLOCKS 8
#define SIDE_BLOCK N/NUM_BLOCKS

__global__ void fast_transpose(const int* a, int* ta, int size) {
  int ix = blockIdx.x*blockDim.x + threadIdx.x;
  int iy = blockIdx.y*blockDim.y + threadIdx.y; 

   __shared__ int tmp[SIDE_BLOCK][SIDE_BLOCK];

  tmp[threadIdx.x][threadIdx.y] = a[ix*size + iy];
  __syncthreads();

  ta[iy*size + ix] = tmp[threadIdx.x][threadIdx.y];

}

int check(const int* a, const int* ta, const int size) {
  int k=0;
  for(int i=0; i<size; i++) {
    for(int j=0; j<size; j++) {
      if(ta[j+i*size] != a[i+j*size]) break;
      else k++;
    }
  }

  if(k==size*size) return 1;
  else return 0;
}


int main() {

  int num_bytes = N*N*sizeof(int);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int *d_a=0, *h_a=0; // device and host pointers
  int *td_a=0, *th_a=0;
  h_a = (int*)malloc(num_bytes);
  th_a = (int*)malloc(num_bytes);
  hipMalloc( (void**)&d_a, num_bytes );
  hipMalloc( (void**)&td_a, num_bytes );

  dim3 grid, block; 
  block.x = NUM_BLOCKS;
  block.y = NUM_BLOCKS;
  grid.x = SIDE_BLOCK; 
  grid.y = SIDE_BLOCK;

  for(int i=0; i<N*N; i++)
    h_a[i] = i+1;

  hipMemcpy(d_a, h_a, num_bytes, hipMemcpyHostToDevice);

  hipEventRecord(start);
  fast_transpose<<<grid, block>>>(d_a, td_a, N);
  hipEventRecord(stop);

  hipMemcpy(th_a,td_a,num_bytes, hipMemcpyDeviceToHost);

  int c = check(th_a,h_a,N);
  printf("correctness: %d\n", c);
  float elapsed_time = 0;
  hipEventElapsedTime(&elapsed_time, start, stop);
  printf("elapsed time: %f\n", elapsed_time);

  /*
  for(int row=0; row<N; row++) { 
    for(int col=0; col<N; col++) {
      printf("%d ", th_a[row*N+col] ); printf("\n");
    }
  }
  */
  
  free( h_a ); free(th_a);
  hipFree( d_a ); hipFree(td_a);

  return 0;
}
