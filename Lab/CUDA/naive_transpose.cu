
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void transpose(int* a, int* ta, int size) {
  int ix = blockIdx.x*blockDim.x + threadIdx.x;
  int iy = blockIdx.y*blockDim.y + threadIdx.y; 

  ta[ix+iy*size] = a[iy+ix*size];
}


int main() {

  int size = 16;
  int num_bytes = size*size*sizeof(int);

  int *d_a=0, *h_a=0; // device and host pointers
  int *td_a=0, *th_a=0;
  h_a = (int*)malloc(num_bytes);
  th_a = (int*)malloc(num_bytes);
  hipMalloc( (void**)&d_a, num_bytes );
  hipMalloc( (void**)&td_a, num_bytes );

  dim3 grid, block; 
  block.x = 4;
  block.y = 4;
  grid.x = size / block.x; 
  grid.y = size / block.y;

  for(int i=0; i<size*size; i++)
    h_a[i] = i+1;

  hipMemcpy(d_a, h_a, num_bytes, hipMemcpyHostToDevice);

  transpose<<<grid, block>>>(d_a, td_a, size);
  hipMemcpy(th_a,td_a,num_bytes, hipMemcpyDeviceToHost);

  for(int row=0; row<size; row++) { 
    for(int col=0; col<size; col++) {
      printf("%d ", th_a[row*size+col] ); printf("\n");
    }
  }

  free( h_a ); free(th_a);
  hipFree( d_a ); hipFree(td_a);

  return 0;
}

