
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 8192
#define BLOCK_DIM_X 8
#define BLOCK_DIM_Y 8
#define NUM_BLOCKS_X N/BLOCK_DIM_X
#define NUM_BLOCKS_Y N/BLOCK_DIM_Y

__global__ void fast_transpose(const int* a, int* ta, int size) {
  int ix = blockIdx.x*blockDim.x + threadIdx.x;
  int iy = blockIdx.y*blockDim.y + threadIdx.y; 

   __shared__ int tmp[BLOCK_DIM_Y][BLOCK_DIM_X];

  tmp[threadIdx.y][threadIdx.x] = a[ix*size + iy];
  __syncthreads();

  ta[iy*size + ix] = tmp[threadIdx.y][threadIdx.x];

}

int check(const int* a, const int* ta, const int size) {
  int k=0;
  for(int i=0; i<size; i++) {
    for(int j=0; j<size; j++) {
      if(ta[j+i*size] != a[i+j*size]) break;
      else k++;
    }
  }

  if(k==size*size) return 1;
  else return 0;
}



int main() {

  int num_bytes = N*N*sizeof(int);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int *d_a=0, *h_a=0; // device and host pointers
  int *td_a=0, *th_a=0;
  h_a = (int*)malloc(num_bytes);
  th_a = (int*)malloc(num_bytes);
  hipMalloc( (void**)&d_a, num_bytes );
  hipMalloc( (void**)&td_a, num_bytes );

  dim3 grid, block; 
  block.x = BLOCK_DIM_X;
  block.y = BLOCK_DIM_Y;
  grid.x = NUM_BLOCKS_X; 
  grid.y = NUM_BLOCKS_Y;

  for(int i=0; i<N*N; i++)
    h_a[i] = i+1;

  hipMemcpy(d_a, h_a, num_bytes, hipMemcpyHostToDevice);

  hipEventRecord(start);
  fast_transpose<<<grid, block>>>(d_a, td_a, N);
  hipEventRecord(stop);

  hipMemcpy(th_a,td_a,num_bytes, hipMemcpyDeviceToHost);

  int c = check(th_a,h_a,N);
  printf("correctness: %d\n", c);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("elapsed time: %f\n", milliseconds);

  free( h_a ); free(th_a);
  hipFree( d_a ); hipFree(td_a);

  return 0;
}
