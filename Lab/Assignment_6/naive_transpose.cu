
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 8192

__global__ void transpose(const int* a, int* ta, const int size) {
  int ix = blockIdx.x*blockDim.x + threadIdx.x;
  int iy = blockIdx.y*blockDim.y + threadIdx.y; 

  ta[iy+ix*size] = a[ix+iy*size];
}

int check(const int* a, const int* ta, const int size) {
  int k=0;
  for(int i=0; i<size; i++) {
    for(int j=0; j<size; j++) {
      if(ta[j+i*size] != a[i+j*size]) break;
      else k++;
    }
  }
  if(k==size*size) return 1;
  else return 0;
}



int main() {

  int num_bytes = N*N*sizeof(int);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int *d_a=0, *h_a=0; // device and host pointers
  int *td_a=0, *th_a=0;
  h_a = (int*)malloc(num_bytes);
  th_a = (int*)malloc(num_bytes);
  hipMalloc( (void**)&d_a, num_bytes );
  hipMalloc( (void**)&td_a, num_bytes );

  dim3 grid, block; 
  block.x = 8;
  block.y = 8;
  grid.x = N / block.x; 
  grid.y = N / block.y;

  for(int i=0; i<N*N; i++)
    h_a[i] = i+1;

  hipMemcpy(d_a, h_a, num_bytes, hipMemcpyHostToDevice);

  hipEventRecord(start);
  transpose<<<grid, block>>>(d_a, td_a, N);
  hipEventRecord(stop);

  hipMemcpy(th_a,td_a,num_bytes, hipMemcpyDeviceToHost);

  int c = check(th_a,h_a,N);
  printf("correctness: %d\n", c);

  free( h_a ); free(th_a);
  hipFree( d_a ); hipFree(td_a);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("elapsed time: %f\n", milliseconds);

  return 0;
}

